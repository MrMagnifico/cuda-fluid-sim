#include "hip/hip_runtime.h"
#include "field_edit.cuh"

DISABLE_WARNINGS_PUSH()
#include <glm/common.hpp>
DISABLE_WARNINGS_POP()


template<typename T>
__global__ void set_source(T* sources, uint2 coords, T val, uint2 field_extents) {
    unsigned int offset = coords.x + (coords.y * field_extents.x);
    sources[offset]     = val;
}

template<typename T>
__global__ void update_field(T* field, T value, uint2 field_extents, uint2 top_left, uint2 bottom_right,
                             UpdateType update_type, bool clampToZero) {
    // Map thread to field coordinates
    unsigned int threadIdX  = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadIdY  = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int updateX    = threadIdX + top_left.x;
    unsigned int updateY    = threadIdY + top_left.y;

    // Threads falling within selection box update their respective cell in the field
    if (updateX <= bottom_right.x && updateY <= bottom_right.y) {
        unsigned int verticalStride = field_extents.x + 2U;
        unsigned int offset         = (updateX + 1U) + (updateY + 1U) * verticalStride; // +1 accounts for ghost cells

        switch (update_type) {
            case Add:       { field[offset] += value; } break;
            case Remove:    {
                field[offset] -= value;
                if (clampToZero) { field[offset] = glm::max(field[offset], T(0.0f)); }
            } break;
        }
    }
}

template<typename T>
__global__ void copyOldField(T* old_field, T* new_field, uint2 old_field_extents, uint2 new_field_extents) {
    unsigned int threadIdX  = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadIdY  = threadIdx.y + blockIdx.y * blockDim.y;
    uint2 smallestExtents   = make_uint2(min(old_field_extents.x, new_field_extents.x),
                                         min(old_field_extents.y, new_field_extents.y));

    // Only copy over values if the thread is operating on an interior cell within the smaller of the two bounds
    if (1U <= threadIdX && threadIdX <= smallestExtents.x &&
        1U <= threadIdY && threadIdY <= smallestExtents.y) {
        unsigned int oldFieldOffset = threadIdX + threadIdY * (old_field_extents.x + 2U);
        unsigned int newFieldOffset = threadIdX + threadIdY * (new_field_extents.x + 2U);
        new_field[newFieldOffset]   = old_field[oldFieldOffset];
    }
}
