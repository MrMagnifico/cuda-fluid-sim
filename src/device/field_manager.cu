#include "hip/hip_runtime.h"
#include "field_manager.cuh"

#include <device/fluid_sim.cuh>
#include <device/fluid_sim.cu>
#include <device/gl_interop.cuh>
#include <device/gl_interop.cu>
#include <device/sources.cuh>
#include <device/sources.cu>
#include <utils/constants.h>
#include <utils/cuda_utils.cuh>

#include <array>
#include <cuda_gl_interop.h>

FieldManager::FieldManager(const RenderConfig& renderConfig, const uint2 fieldExtents,
                           const GLuint sourcesDensityTex, const GLuint densitiesTex,
                           const GLuint sourcesVelocityTex, const GLuint velocitiesTex)
    : m_fieldExtents(fieldExtents)
    , m_renderConfig(renderConfig) {
    // Calculate memory sizes and dimensions
    m_paddedfieldExtents    = uint2(fieldExtents.x + 2UL, fieldExtents.y + 2UL);
    m_gridDims              = dim3((m_paddedfieldExtents.x / utils::BLOCK_SIZE.x) + 1U, (m_paddedfieldExtents.y / utils::BLOCK_SIZE.y) + 1U);   // Grid dimensions needed for workload distribution

    // Allocate and zero initialise memory for fields
    std::array<glm::vec3**, utils::FIELDS_PER_TYPE> fieldsDensity   = { &m_densitySources, &m_densities, &m_densitiesPrev };
    std::array<glm::vec2**, utils::FIELDS_PER_TYPE> fieldsVelocity  = { &m_velocitySources, &m_velocities, &m_velocitiesPrev };
    size_t fieldSizeDensity                                         = (m_paddedfieldExtents.x) * (m_paddedfieldExtents.y) * sizeof(glm::vec3); // Account for boundaries
    size_t fieldSizeVelocity                                        = (m_paddedfieldExtents.x) * (m_paddedfieldExtents.y) * sizeof(glm::vec2); // Account for boundaries
    for (size_t fieldIdx = 0UL; fieldIdx < utils::FIELDS_PER_TYPE; fieldIdx++) {
        auto densityField   = fieldsDensity[fieldIdx];
        auto velocityField  = fieldsVelocity[fieldIdx];
        CUDA_ERROR(hipMalloc(densityField, fieldSizeDensity));
        CUDA_ERROR(hipMalloc(velocityField, fieldSizeVelocity));
        CUDA_ERROR(hipMemset(*densityField, 0, fieldSizeDensity));
        CUDA_ERROR(hipMemset(*velocityField, 0, fieldSizeVelocity));
    }

    // Create OpenGL textures resource handles
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_sourcesDensityResource,   sourcesDensityTex,  GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_densitiesResource,        densitiesTex,       GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_sourcesVelocityResource,  sourcesVelocityTex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_velocitiesResource,       velocitiesTex,      GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
}

void FieldManager::copyFieldsToTextures() {
    hipSurfaceObject_t sourcesDensitySurface   = utils::createSurfaceFromTextureResource(m_sourcesDensityResource);
    hipSurfaceObject_t densitiesSurface        = utils::createSurfaceFromTextureResource(m_densitiesResource);
    hipSurfaceObject_t sourcesVelocitySurface  = utils::createSurfaceFromTextureResource(m_sourcesVelocityResource);
    hipSurfaceObject_t velocitiesSurface       = utils::createSurfaceFromTextureResource(m_velocitiesResource);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densitySources,     sourcesDensitySurface,  m_fieldExtents);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densities,          densitiesSurface,       m_fieldExtents);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocities,         velocitiesSurface,      m_fieldExtents);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocitySources,    sourcesVelocitySurface, m_fieldExtents);
    CUDA_ERROR(hipDeviceSynchronize()); // Ensure that copying is over before terminating resource handles
    CUDA_ERROR(hipDestroySurfaceObject(sourcesDensitySurface));
    CUDA_ERROR(hipDestroySurfaceObject(densitiesSurface));
    CUDA_ERROR(hipDestroySurfaceObject(sourcesVelocitySurface));
    CUDA_ERROR(hipDestroySurfaceObject(velocitiesSurface));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_sourcesDensityResource));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_densitiesResource));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_sourcesVelocityResource));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_velocitiesResource));
}

void FieldManager::setSourceDensity(uint2 coords, glm::vec3 val) { set_source<<<1, 1>>>(m_densitySources, coords, val, m_paddedfieldExtents); }

void FieldManager::setSourceVelocity(uint2 coords, glm::vec2 val) { set_source<<<1, 1>>>(m_velocitySources, coords, val, m_paddedfieldExtents); }

void FieldManager::simulate() {
    // velocityStep();
    densityStep();
}

void FieldManager::densityStep() {
    add_sources<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densities, m_densitySources, m_renderConfig.timeStep, m_paddedfieldExtents.x * m_paddedfieldExtents.y);
    std::swap(m_densities, m_densitiesPrev);
    size_t sharedMemSize = (utils::BLOCK_SIZE.x + 2UL) * (utils::BLOCK_SIZE.y + 2UL) * sizeof(glm::vec3) * 2UL; // Account for ghost cells and the fact that we store TWO fields (old and new)
    diffuse<<<m_gridDims, utils::BLOCK_SIZE, sharedMemSize>>>(m_densitiesPrev, m_densities,
                                                              m_fieldExtents, m_paddedfieldExtents.x * m_paddedfieldExtents.y,
                                                              Conserve, m_renderConfig.timeStep, m_renderConfig.diffusionRate, m_renderConfig.diffusionSimSteps);
    // std::swap(m_densities, m_densitiesPrev);
    // advect<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densitiesPrev, m_densities, m_velocities,
    //                                           m_fieldExtents, m_paddedfieldExtents.x * m_paddedfieldExtents.y,
    //                                           Conserve, m_renderConfig.timeStep);
}

void FieldManager::velocityStep() {
    add_sources<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocities, m_velocitySources, m_renderConfig.timeStep, m_paddedfieldExtents.x * m_paddedfieldExtents.y);
    std::swap(m_velocities, m_velocitiesPrev);
    size_t sharedMemSize = (utils::BLOCK_SIZE.x + 2UL) * (utils::BLOCK_SIZE.y + 2UL) * sizeof(glm::vec2) * 2UL; // Account for ghost cells and the fact that we store TWO fields (old and new)
    diffuse<<<m_gridDims, utils::BLOCK_SIZE, sharedMemSize>>>(m_velocitiesPrev, m_velocities,
                                                              m_fieldExtents, m_paddedfieldExtents.x * m_paddedfieldExtents.y,
                                                              Conserve, m_renderConfig.timeStep, m_renderConfig.diffusionRate, m_renderConfig.diffusionSimSteps);
    std::swap(m_velocities, m_velocitiesPrev);
    advect<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densitiesPrev, m_densities, m_velocities,
                                              m_fieldExtents, m_paddedfieldExtents.x * m_paddedfieldExtents.y,
                                              Conserve, m_renderConfig.timeStep);
    // TODO: Add projection step
}
