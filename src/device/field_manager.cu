#include "hip/hip_runtime.h"
#include "field_manager.cuh"

#include <device/fluid_sim.cuh>
#include <device/gl_interop.cuh>
#include <device/sources.cuh>
#include <utils/constants.h>
#include <utils/cuda_utils.cuh>

#include <array>

FieldManager::FieldManager(const uint2 fieldExtents, const RenderConfig& renderConfig)
    : m_fieldExtents(fieldExtents)
    , m_renderConfig(renderConfig) {
    // Calculate memory sizes and dimensions
    m_paddedfieldExtents    = uint2(fieldExtents.x + 2UL, fieldExtents.y + 2UL);
    m_fieldsSize            = (m_paddedfieldExtents.x)      * (m_paddedfieldExtents.y)      * sizeof(glm::vec3);                                // Account for boundaries
    m_sharedMemSize         = (utils::BLOCK_SIZE.x + 2UL)   * (utils::BLOCK_SIZE.y + 2UL)   * sizeof(glm::vec3) * 2UL;                          // Account for ghost cells and the fact that we store TWO fields (old and new)
    m_gridDims              = dim3((m_paddedfieldExtents.x / utils::BLOCK_SIZE.x) + 1U, (m_paddedfieldExtents.y / utils::BLOCK_SIZE.y) + 1U);   // Grid dimensions needed for workload distribution

    std::array<glm::vec3**, 3UL> fields = { &m_sources, 
                                            &m_densities, &m_densitiesPrev };
    for (auto field : fields) {
        CUDA_ERROR(hipMalloc(field, m_fieldsSize));
        CUDA_ERROR(hipMemset(*field, 0, m_fieldsSize));
    }
}

FieldManager::~FieldManager() {
    std::array<glm::vec3*, 3UL> fields = { m_sources, 
                                           m_densities, m_densitiesPrev };
    for (auto field : fields) { CUDA_ERROR(hipFree(field)); }
}

void FieldManager::copyFieldsToTexture(hipSurfaceObject_t sourcesSurface, hipSurfaceObject_t densitiesSurface) {
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densities, densitiesSurface, m_fieldExtents);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_sources, sourcesSurface, m_fieldExtents);    
}

void FieldManager::setSource(uint2 coords, glm::vec3 val) { set_source<<<1, 1>>>(m_sources, coords, val, m_paddedfieldExtents); }

void FieldManager::simulate() {
    add_sources<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densities, m_sources, m_renderConfig.timeStep, m_paddedfieldExtents.x * m_paddedfieldExtents.y);
    std::swap(m_densities, m_densitiesPrev);
    diffuse<<<m_gridDims, utils::BLOCK_SIZE, m_sharedMemSize>>>(m_densitiesPrev, m_densities, m_fieldExtents, m_paddedfieldExtents.x * m_paddedfieldExtents.y,
                                                                m_renderConfig.timeStep, m_renderConfig.diffusionRate, m_renderConfig.diffusionSimSteps);
}
