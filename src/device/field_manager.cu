#include "hip/hip_runtime.h"
#include "field_manager.cuh"

#include <device/fluid_sim.cuh>
#include <device/fluid_sim.cu>
#include <device/gl_interop.cuh>
#include <device/gl_interop.cu>
#include <device/sources.cuh>
#include <device/sources.cu>
#include <utils/constants.h>
#include <utils/cuda_utils.cuh>

#include <array>
#include <cuda_gl_interop.h>

FieldManager::FieldManager(const RenderConfig& renderConfig, const uint2 fieldExtents,
                           const GLuint sourcesDensityTex, const GLuint densitiesTex,
                           const GLuint sourcesVelocityTex, const GLuint velocitiesTex)
    : m_fieldExtents(fieldExtents)
    , m_renderConfig(renderConfig) {
    // Calculate memory sizes and dimensions
    m_paddedfieldExtents    = uint2(fieldExtents.x + 2UL, fieldExtents.y + 2UL);
    m_gridDims              = dim3((m_paddedfieldExtents.x / utils::BLOCK_SIZE.x) + 1U, (m_paddedfieldExtents.y / utils::BLOCK_SIZE.y) + 1U);   // Grid dimensions needed for workload distribution

    // Allocate and zero initialise memory for fields
    std::array<glm::vec4**, utils::FIELDS_PER_TYPE> fieldsDensity   = { &m_densitySources, &m_densities, &m_densitiesPrev };
    std::array<glm::vec2**, utils::FIELDS_PER_TYPE> fieldsVelocity  = { &m_velocitySources, &m_velocities, &m_velocitiesPrev };
    size_t fieldSizeDensity                                         = (m_paddedfieldExtents.x) * (m_paddedfieldExtents.y) * sizeof(glm::vec4); // Account for boundaries
    size_t fieldSizeVelocity                                        = (m_paddedfieldExtents.x) * (m_paddedfieldExtents.y) * sizeof(glm::vec2); // Account for boundaries
    for (size_t fieldIdx = 0UL; fieldIdx < utils::FIELDS_PER_TYPE; fieldIdx++) {
        auto densityField   = fieldsDensity[fieldIdx];
        auto velocityField  = fieldsVelocity[fieldIdx];
        CUDA_ERROR(hipMalloc(densityField, fieldSizeDensity));
        CUDA_ERROR(hipMalloc(velocityField, fieldSizeVelocity));
        CUDA_ERROR(hipMemset(*densityField, 0, fieldSizeDensity));
        CUDA_ERROR(hipMemset(*velocityField, 0, fieldSizeVelocity));
    }

    // Allocate and zero initialise memory for intermediate value fields
    std::array<float**, 2UL> fieldsIntermediate = { &m_gradientField, &m_projectionField };
    size_t fieldSizeIntermediate                = (m_paddedfieldExtents.x) * (m_paddedfieldExtents.y) * sizeof(float);
    for (float** field : fieldsIntermediate) {
        CUDA_ERROR(hipMalloc(field, fieldSizeIntermediate));
        CUDA_ERROR(hipMemset(*field, 0, fieldSizeIntermediate));
    }

    // Create OpenGL textures resource handles
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_sourcesDensityResource,   sourcesDensityTex,  GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_densitiesResource,        densitiesTex,       GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_sourcesVelocityResource,  sourcesVelocityTex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    CUDA_ERROR(hipGraphicsGLRegisterImage(&m_velocitiesResource,       velocitiesTex,      GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
}

void FieldManager::copyFieldsToTextures() {
    hipSurfaceObject_t sourcesDensitySurface   = utils::createSurfaceFromTextureResource(m_sourcesDensityResource);
    hipSurfaceObject_t densitiesSurface        = utils::createSurfaceFromTextureResource(m_densitiesResource);
    hipSurfaceObject_t sourcesVelocitySurface  = utils::createSurfaceFromTextureResource(m_sourcesVelocityResource);
    hipSurfaceObject_t velocitiesSurface       = utils::createSurfaceFromTextureResource(m_velocitiesResource);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densitySources,     sourcesDensitySurface,  m_fieldExtents);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densities,          densitiesSurface,       m_fieldExtents);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocities,         velocitiesSurface,      m_fieldExtents);
    copyFieldToTexture<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocitySources,    sourcesVelocitySurface, m_fieldExtents);
    CUDA_ERROR(hipDeviceSynchronize()); // Ensure that copying is over before terminating resource handles
    CUDA_ERROR(hipDestroySurfaceObject(sourcesDensitySurface));
    CUDA_ERROR(hipDestroySurfaceObject(densitiesSurface));
    CUDA_ERROR(hipDestroySurfaceObject(sourcesVelocitySurface));
    CUDA_ERROR(hipDestroySurfaceObject(velocitiesSurface));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_sourcesDensityResource));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_densitiesResource));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_sourcesVelocityResource));
    CUDA_ERROR(hipGraphicsUnmapResources(1, &m_velocitiesResource));
}

void FieldManager::setSourceDensity(uint2 coords, glm::vec4 val) { set_source<<<1, 1>>>(m_densitySources, coords, val, m_paddedfieldExtents); }

void FieldManager::setSourceVelocity(uint2 coords, glm::vec2 val) { set_source<<<1, 1>>>(m_velocitySources, coords, val, m_paddedfieldExtents); }

void FieldManager::simulate() {
    velocityStep();
    densityStep();
}

void FieldManager::densityStep() {
    if (m_renderConfig.densityAddSources) { add_sources<<<m_gridDims, utils::BLOCK_SIZE>>>(m_densities, m_densitySources, m_fieldExtents, m_renderConfig.simulationParams); }
    if (m_renderConfig.densityDiffuse) {
        std::swap(m_densities,      m_densitiesPrev);
        size_t sharedMemSize = (utils::BLOCK_SIZE.x + 2UL) * (utils::BLOCK_SIZE.y + 2UL) * sizeof(glm::vec4) * 2UL; // Account for ghost cells and the fact that we store TWO fields (old and new)
        diffuse<<<m_gridDims, utils::BLOCK_SIZE, sharedMemSize>>>(m_densitiesPrev, m_densities, m_fieldExtents, Conserve, m_renderConfig.simulationParams);
    }
    if (m_renderConfig.densityAdvect) {
        std::swap(m_densities,      m_densitiesPrev);
        advect<glm::vec4, glm::vec2><<<m_gridDims, utils::BLOCK_SIZE>>>(m_densitiesPrev, m_densities, m_velocities, m_fieldExtents,
                                                                        Conserve, m_renderConfig.simulationParams);
    }
}

void FieldManager::velocityStep() {
    if (m_renderConfig.velocityAddSources) { add_sources<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocities, m_velocitySources, m_fieldExtents, m_renderConfig.simulationParams); }
    if (m_renderConfig.velocityDiffuse) {
        std::swap(m_velocities, m_velocitiesPrev);
        size_t sharedMemSize = (utils::BLOCK_SIZE.x + 2UL) * (utils::BLOCK_SIZE.y + 2UL) * sizeof(glm::vec2) * 2UL; // Account for ghost cells and the fact that we store TWO fields (old and new)
        diffuse<<<m_gridDims, utils::BLOCK_SIZE, sharedMemSize>>>(m_velocitiesPrev, m_velocities, m_fieldExtents, Reverse, m_renderConfig.simulationParams);
        if (m_renderConfig.velocityProject) { project<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocities, m_gradientField, m_projectionField,
                                                                                         m_fieldExtents, m_renderConfig.simulationParams); }
    }
    if (m_renderConfig.velocityAdvect) {
        std::swap(m_velocities, m_velocitiesPrev);
        advect<glm::vec2, glm::vec2><<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocitiesPrev, m_velocities, m_velocitiesPrev, m_fieldExtents, 
                                                                        Reverse, m_renderConfig.simulationParams);
        if (m_renderConfig.velocityProject) { project<<<m_gridDims, utils::BLOCK_SIZE>>>(m_velocities, m_gradientField, m_projectionField,
                                                                                         m_fieldExtents, m_renderConfig.simulationParams); }
    }
}
