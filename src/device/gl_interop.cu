#include "hip/hip_runtime.h"
#include "gl_interop.cuh"

#include <device/utils.cuh>

template<typename T>
__global__ void copyFieldToTexture(T* field, hipSurfaceObject_t texture_surface, uint2 texture_extents) {
    unsigned int tidX   = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int tidY   = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidX < texture_extents.x && tidY < texture_extents.y) { 
        unsigned int surfaceX   = tidX + 1U;
        unsigned int surfaceY   = tidY + 1U;
        unsigned int offset     = surfaceX + surfaceY * (texture_extents.x + 2U);
        T value                 = field[offset];

        // Use float4 as the expected texture format is GL_RGBA32F
        float4 valuePadded = toRGBA(value);
        surf2Dwrite(valuePadded, texture_surface, tidX * sizeof(float4), tidY);
    }
}
