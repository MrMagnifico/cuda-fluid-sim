#include "cuda_utils.cuh"

#include <cstdio>

inline void utils::HandleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "\n%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

hipSurfaceObject_t utils::createSurfaceFromTextureResource(hipGraphicsResource_t textureResource) {
    CUDA_ERROR(hipGraphicsMapResources(1, &textureResource));
    hipArray_t textureArray;
    CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&textureArray, textureResource, 0, 0));
    hipResourceDesc textureResourceDescriptor;
    textureResourceDescriptor.resType           = hipResourceTypeArray;
    textureResourceDescriptor.res.array.array   = textureArray;
    hipSurfaceObject_t textureSurface;
    CUDA_ERROR(hipCreateSurfaceObject(&textureSurface, &textureResourceDescriptor));
    return textureSurface;
}
