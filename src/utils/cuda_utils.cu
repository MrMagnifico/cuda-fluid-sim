#include "cuda_utils.cuh"


hipSurfaceObject_t utils::createSurfaceFromTextureResource(hipGraphicsResource_t textureResource) {
    CUDA_ERROR(hipGraphicsMapResources(1, &textureResource));
    hipArray_t textureArray;
    CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&textureArray, textureResource, 0, 0));
    hipResourceDesc textureResourceDescriptor;
    textureResourceDescriptor.resType           = hipResourceTypeArray;
    textureResourceDescriptor.res.array.array   = textureArray;
    hipSurfaceObject_t textureSurface;
    CUDA_ERROR(hipCreateSurfaceObject(&textureSurface, &textureResourceDescriptor));
    return textureSurface;
}
